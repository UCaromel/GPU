
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <chrono>
#include <fstream>
#include <iomanip>

// Fonction à intégrer
__device__ double function(double x) {
    return 4.0 / (1.0 + x * x);
}

// Kernel CUDA pour l'évaluation de l'intégrale avec la règle de Simpson composée
__global__ void integrate(double* result, int num_subintervals, double dx) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    double sum = 0.0;
    for (int i = tid; i < num_subintervals; i += blockDim.x * gridDim.x) {
        double x0 = i * dx;
        double x1 = (i + 1) * dx;
        double x_mid = (x0 + x1) / 2.0;
        sum += function(x0) + 4.0 * function(x_mid) + function(x1);
    }
    
    result[tid] = sum * dx / 6.0;
}

int main() {
    const int num_subintervals_start = 2;
    const int num_subintervals_end = 1073741825;
    const int num_threads_per_block = 64;
    const int num_blocks = 3584;
    const double a = 0.0; // Borne inférieure
    const double b = 1.0; // Borne supérieure

    std::ofstream error_file("error_cuda.txt");
    std::ofstream time_file("time_cuda.txt");

    for (int num_subintervals = num_subintervals_start; num_subintervals <= num_subintervals_end; num_subintervals *= 2) {
        // Allocation mémoire sur le CPU pour stocker les résultats
        double* result_cpu = new double[num_subintervals];

        // Allocation mémoire sur le GPU
        double* result_gpu;
        hipMalloc((void**)&result_gpu, num_subintervals * sizeof(double));

        // Paramètres du GPU

        // Mesurer le temps d'exécution
        auto start_time = std::chrono::high_resolution_clock::now();

        // Appeler le kernel CUDA pour l'évaluation de l'intégrale
        integrate<<<num_blocks, num_threads_per_block>>>(result_gpu, num_subintervals, (b - a) / num_subintervals);

        // Copier les résultats du GPU vers le CPU
        hipMemcpy(result_cpu, result_gpu, num_subintervals * sizeof(double), hipMemcpyDeviceToHost);

        // Calculer le résultat final en additionnant les résultats de chaque thread
        double final_result = 0.0;
        for (int i = 0; i < num_subintervals; ++i) {
            final_result += result_cpu[i];
        }

        // Mesurer le temps total d'exécution
        auto end_time = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> duration = end_time - start_time;
        double execution_time = duration.count();

        // Calculer l'erreur par rapport à la valeur exacte
        double pi = 3.141592653589793238462643383279502884197;
        double error = std::abs(final_result - pi);

        // Enregistrement des résultats dans les fichiers
        error_file << num_subintervals << " " << error << std::endl;
        time_file << num_subintervals << " " << execution_time << std::endl;

        // Libérer la mémoire
        delete[] result_cpu;
        hipFree(result_gpu);
    }

    error_file.close();
    time_file.close();

    return 0;
}
