
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <vector>
#include <fstream>
#include <chrono>

const double PI = 3.14159265358979323846;  // Utiliser une valeur explicite pour PI

// Fonction représentant la solution exacte de l'équation de la chaleur
double exactSolution(double alpha, double x, double t) {
    return 1.0 / sqrt(4.0 * PI * alpha * t) * exp(-x * x / (4.0 * alpha * t));
}

// Fonction représentant l'équation de la chaleur
__device__ double heatEquation(double alpha, const double* u, int i) {
    return alpha * (u[i - 1] - 2 * u[i] + u[i + 1]);
}

// Kernel CUDA pour la résolution de l'équation de la chaleur avec RK4
__global__ void rungeKuttaHeatEquation(double alpha, double* u, double h, double endTime, int numPoints) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i > 0 && i < numPoints - 1) {
        // Copie temporaire de la solution actuelle
        double tempU[3] = { u[i - 1], u[i], u[i + 1] };

        // Calcul des coefficients k1, k2, k3, k4
        double k1 = h * heatEquation(alpha, tempU, 1);
        double k2 = h * heatEquation(alpha, tempU, 1);
        double k3 = h * heatEquation(alpha, tempU, 1);
        double k4 = h * heatEquation(alpha, tempU, 1);

        // Mise à jour de la solution
        u[i] = tempU[1] + (1.0 / 6.0) * (k1 + 2.0 * k2 + 2.0 * k3 + k4);
    }
}

int main() {
    // Paramètres
    double alpha = 0.01;  // constante de diffusion thermique
    double endTime = 0.1; // temps final

    // Itérations sur différents maillages
    std::vector<int> mesh_sizes = {2, 4, 8, 16, 32, 64, 128, 256, 512, 1024, 2048, 4096, 8192, 16384, 32768, 65536, 131072, 262144, 524288, 1048576, 2097152, 4194304};
    // Fichier de sortie pour les erreurs
    std::ofstream errorFile("error_cuda.txt");
    // Fichier de sortie pour les temps d'exécution
    std::ofstream timeFile("time_cuda.txt");

    for (int mesh_size : mesh_sizes) {
        // Calcul de la solution exacte
        std::vector<double> exact_solution(mesh_size, 0.0);
        for (int i = 0; i < mesh_size; ++i) {
            double x = i * (1.0 / static_cast<double>(mesh_size));
            exact_solution[i] = exactSolution(alpha, x, endTime);
        }

        // Pas de discrétisation en espace
        double h = 1.0 / static_cast<double>(mesh_size);

        // Conditions initiales
        std::vector<double> numerical_solution(mesh_size, 0.0);

        // Initialiser la condition initiale (une gaussienne centrée)
        for (int i = 0; i < mesh_size; ++i) {
            double x = i * h;
            numerical_solution[i] = exp(-x * x / (4.0 * alpha));
        }

        // Transfert des données vers le périphérique CUDA
        double* d_numerical_solution;
        hipMalloc(&d_numerical_solution, mesh_size * sizeof(double));
        hipMemcpy(d_numerical_solution, numerical_solution.data(), mesh_size * sizeof(double), hipMemcpyHostToDevice);

        // Chronométrage du kernel CUDA
        auto start = std::chrono::high_resolution_clock::now();

        // Résolution de l'équation de la chaleur avec RK4 en parallèle avec CUDA
        for (double t = 0.0; t < endTime; t += h) {
            rungeKuttaHeatEquation<<<(mesh_size + 63) / 64, 64>>>(alpha, d_numerical_solution, h, endTime, mesh_size);
            hipDeviceSynchronize();
        }

        auto end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> duration = end - start;

        // Transfert des résultats depuis le périphérique CUDA
        hipMemcpy(numerical_solution.data(), d_numerical_solution, mesh_size * sizeof(double), hipMemcpyDeviceToHost);

        // Libération de la mémoire du périphérique CUDA
        hipFree(d_numerical_solution);

        // Calcul de l'erreur relative moyenne
        double total_error = 0.0;
        for (int i = 0; i < mesh_size; ++i) {
            total_error += std::abs(numerical_solution[i] - exact_solution[i]);
        }
        double mean_relative_error = total_error / static_cast<double>(mesh_size);

        // Écriture des résultats dans les fichiers de sortie
        errorFile << mesh_size << " " << mean_relative_error << std::endl;
        timeFile << mesh_size << " " << duration.count() << std::endl;
    }

    // Fermeture des fichiers de sortie
    errorFile.close();
    timeFile.close();

    return 0;
}
