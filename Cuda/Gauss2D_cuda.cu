
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <chrono>
#include <fstream>

// Fonction à intégrer
__device__ double function(double x) {
    return 4.0 / (1.0 + x * x);
}

// Kernel CUDA pour l'évaluation de l'intégrale avec la méthode de Gauss
__global__ void integrate(double* result, int num_subintervals, double dx) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    double sum = 0.0;
    for (int i = tid; i < num_subintervals; i += blockDim.x * gridDim.x) {
        double x0 = i * dx;
        double x1 = (i + 1) * dx;

        // Points de quadrature et poids pour la méthode de Gauss (ici, 4 points de Gauss)
        double xi[4] = {-0.861136, -0.339981, 0.339981, 0.861136};
        double w[4] = {0.347855, 0.652145, 0.652145, 0.347855};

        for (int j = 0; j < 4; ++j) {
            double x_quad = 0.5 * ((x1 - x0) * xi[j] + x1 + x0);
            sum += w[j] * function(x_quad);
        }
    }
    
    result[tid] = sum * 0.5 * dx;
}

int main() {
    const int num_subintervals_start = 1000;
    const int num_subintervals_end = 1000000000;
    const int num_threads_per_block_values[] = {128, 256, 512, 1024};
    const int num_values = sizeof(num_threads_per_block_values) / sizeof(num_threads_per_block_values[0]);
    const double a = 0.0; // Borne inférieure
    const double b = 1.0; // Borne supérieure

    std::ofstream error_file("error_results.txt");
    std::ofstream time_file("time_results.txt");

    for (int num_threads_idx = 0; num_threads_idx < num_values; ++num_threads_idx) {
        int num_threads_per_block = num_threads_per_block_values[num_threads_idx];

        error_file << "Threads per block: " << num_threads_per_block << std::endl;
        time_file << "Threads per block: " << num_threads_per_block << std::endl;

        for (int num_subintervals = num_subintervals_start; num_subintervals <= num_subintervals_end; num_subintervals *= 2) {
            // Allocation mémoire sur le CPU pour stocker les résultats
            double* result_cpu = new double[num_subintervals];

            // Allocation mémoire sur le GPU
            double* result_gpu;
			hipMalloc((void**)&result_gpu, num_subintervals * sizeof(double));

			// Initialize GPU memory
			hipMemset(result_gpu, 0, num_subintervals * sizeof(double));

            // Paramètres du GPU
            const int num_blocks = std::min(60, (num_subintervals + num_threads_per_block - 1) / num_threads_per_block);

            // Mesurer le temps d'exécution
            auto start_time = std::chrono::high_resolution_clock::now();

            // Appeler le kernel CUDA pour l'évaluation de l'intégrale
            integrate<<<num_blocks, num_threads_per_block>>>(result_gpu, num_subintervals, (b - a) / num_subintervals);

            // Copier les résultats du GPU vers le CPU
            hipMemcpy(result_cpu, result_gpu, num_subintervals * sizeof(double), hipMemcpyDeviceToHost);

            // Calculer le résultat final en additionnant les résultats de chaque thread
            double final_result = 0.0;
            for (int i = 0; i < num_subintervals; ++i) {
                final_result += result_cpu[i];
            }

            // Mesurer le temps total d'exécution
            auto end_time = std::chrono::high_resolution_clock::now();
            std::chrono::duration<double> duration = end_time - start_time;
            double execution_time = duration.count();

            // Calculer l'erreur par rapport à la valeur exacte
            double error = std::abs(final_result - 3.141592653589793238462643383279502884197);

            // Enregistrement des résultats dans les fichiers
            error_file << num_subintervals << " " << error << std::endl;
            time_file << num_subintervals << " " << execution_time << std::endl;

            // Libérer la mémoire
            delete[] result_cpu;
            hipFree(result_gpu);
        }
    }

    error_file.close();
    time_file.close();

    return 0;
}
