#include "hip/hip_runtime.h"
#include <iostream>
#include "eigen-3.4.0\eigen-3.4.0\Eigen\Dense"
#include <fstream>
#include <iomanip>
#include <chrono>
#include <string>

#include <hiprand/hiprand_kernel.h>

using namespace Eigen;
using namespace std;
using namespace std::chrono;

__device__ double f(double x, double y) {
    return x * y * cos(x) * sin(2 * y);
}

__global__ void computeGauss2DPointsWeights(double* points, double* weights, int numPoints, int totalPoints) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < totalPoints) {
        int i = tid / numPoints;
        int j = tid % numPoints;

        double xi = -1.0 + 2.0 * (i + 0.5) / numPoints;
        double eta = -1.0 + 2.0 * (j + 0.5) / numPoints;
        double wi = 2.0 / numPoints;

        points[tid * 2] = xi;
        points[tid * 2 + 1] = eta;
        weights[tid] = wi;
    }
}

__global__ void gauss2DIntegrationKernel(double* result, double* points, double* weights, double a1, double b1, double a2, double b2, int numPoints, int totalPoints) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < totalPoints) {
        int i = tid / numPoints;
        int j = tid % numPoints;

        double xi = points[tid * 2];
        double eta = points[tid * 2 + 1];

        result[tid] = weights[i] * weights[j] * f((xi + 1) / 2.0 * (b1 - a1) + a1, (eta + 1) / 2.0 * (b2 - a2) + a2);
    }
}

double gauss2DIntegrationCUDA(double a1, double b1, double a2, double b2, int numPoints) {
    int totalPoints = numPoints * numPoints;

    // Allocate device memory
    double* d_points;
    double* d_weights;
    double* d_result;

    hipMalloc((void**)&d_points, totalPoints * 2 * sizeof(double));
    hipMalloc((void**)&d_weights, totalPoints * sizeof(double));
    hipMalloc((void**)&d_result, totalPoints * sizeof(double));

    // Launch CUDA kernel to compute points and weights
    int threadsPerBlock = 1024;
    int numBlocks = (totalPoints + threadsPerBlock - 1) / threadsPerBlock;

    computeGauss2DPointsWeights<<<numBlocks, threadsPerBlock>>>(d_points, d_weights, numPoints, totalPoints);
    hipDeviceSynchronize();

    // Launch CUDA kernel for integration
    gauss2DIntegrationKernel<<<numBlocks, threadsPerBlock>>>(d_result, d_points, d_weights, a1, b1, a2, b2, numPoints, totalPoints);
    hipDeviceSynchronize();

    // Copy results from device to host
    double* h_result = new double[totalPoints];
    hipMemcpy(h_result, d_result, totalPoints * sizeof(double), hipMemcpyDeviceToHost);

    // Calculate the final result on the CPU
    double final_result = 0.0;
    for (int i = 0; i < totalPoints; ++i) {
        final_result += h_result[i];
    }

    // Free allocated memory
    delete[] h_result;
    hipFree(d_points);
    hipFree(d_weights);
    hipFree(d_result);

    // Calculate the average and integrate over the area
    double area = (b1 - a1) * (b2 - a2);
    double integral = final_result * 0.25 * area;

    return integral;
}

void performComputation(int numPoints) {
    double a1 = 0.0, b1 = 10.0;
    double a2 = 0.0, b2 = 10.0;

    auto start_time = high_resolution_clock::now();
    double result = gauss2DIntegrationCUDA(a1, b1, a2, b2, numPoints);
    auto end_time = high_resolution_clock::now();

    auto duration = duration_cast<milliseconds>(end_time - start_time).count() / 1000.0;

    double ex = 13.1913267088667;
    double error = abs(result - ex);

    std::string errorFilename = "error_cuda.txt";
    std::string timeFilename = "time_cuda.txt";

    std::ofstream errorFile(errorFilename, std::ios_base::app);  
    std::ofstream timeFile(timeFilename, std::ios_base::app);    

    if (errorFile.is_open() && timeFile.is_open()) {
        errorFile << std::setprecision(20) << numPoints << " " << error << std::endl;
        timeFile << std::setprecision(20) << numPoints << " " << duration << std::endl;

        errorFile.close();
        timeFile.close();
    } else {
        std::cerr << "Error: Unable to open files for writing." << std::endl;
    }
}

int main() {
    int maxExponent = 14; // 2^30

    for (int exp = 1; exp <= maxExponent; ++exp) {
        int numPoints = pow(2, exp);

        performComputation(numPoints);
    }

    return 0;
}
